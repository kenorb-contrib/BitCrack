#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <>
#include "KeySearchTypes.h"
#include "CKSD.h"
#include "ptx.cuh"
#include "secp256k1.cuh"

#include "sha256.cuh"
#include "ripemd160.cuh"

#include "secp256k1.h"

#include "CudaHashLookup.cuh"
#include "CudaAtomicList.cuh"
#include "CudaDeviceKeys.cuh"

__constant__ unsigned int _INC_X[8];

__constant__ unsigned int _INC_Y[8];
